#include "hip/hip_runtime.h"
// CIS565 CUDA Raytracer: A parallel raytracer for Patrick Cozzi's CIS565: GPU Computing at the University of Pennsylvania
// Written by Yining Karl Li, Copyright (c) 2012 University of Pennsylvania
// This file includes code from:
//       Rob Farber for CUDA-GL interop, from CUDA Supercomputing For The Masses: http://www.drdobbs.com/architecture-and-design/cuda-supercomputing-for-the-masses-part/222600097
//       Peter Kutz and Yining Karl Li's GPU Pathtracer: http://gpupathtracer.blogspot.com/
//       Yining Karl Li's TAKUA Render, a massively parallel pathtracing renderer: http://www.yiningkarlli.com

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <cmath>
#include "sceneStructs.h"
#include "glm/glm.hpp"
#include "glm/gtc/matrix_transform.hpp"
#include "utilities.h"
#include "raytraceKernel.h"
#include "intersections.h"
#include "interactions.h"
#include <vector>

// Optimizations and add ons
#define JITTER 1
#define COMPACTION 1
#define ACCUMULATION 1
#define DOF 1
#define FRESNEL 1
#define SCHLICK 0
#define PATHTRACER 0	
#define PAINTERLY 0
#define PHOTONMAP 1

#if CUDA_VERSION >= 5000
    #include <hip/hip_vector_types.h>
#else
    #include <cutil_math.h>
#endif

enum {
	DISP_RAYTRACE,
	DISP_PHOTONS,
	DISP_GATHER,
	DISP_COMBINED,
	DISP_PATHTRACE,
	DISP_TOTAL
};


#if PHOTONMAP
int numPhotons = 10000;

int numBounces = 5;			//hard limit of 3 bounces for now
float totalEnergy = 80;			//total amount of energy in the scene, used for calculating flux per photon


photon* cudaPhotonPool;		//global variable of photons
glm::vec3* cudaPhotonMapImage;

#define RADIUS 1.5

#endif

glm::vec3* accumulatorImage = NULL;
extern bool singleFrameMode;
extern int mode;

//scene data
glm::vec3* cudaimage;
staticGeom* cudageoms;
material* cudamaterials;
int* cudaLights;
float* cudaAccumLightProbabilities;
int numLights;
int numGeoms;

void checkCUDAError(const char *msg) {
  hipError_t err = hipGetLastError();
  if( hipSuccess != err) {
    fprintf(stderr, "Cuda error: %s: %s.\n", msg, hipGetErrorString( err) ); 
	getchar();
    exit(EXIT_FAILURE); 
  }
} 

//LOOK: This function demonstrates how to use thrust for random number generation on the GPU!
//Function that generates static.
__host__ __device__ glm::vec3 generateRandomNumberFromThread(glm::vec2 resolution, float time, int x, int y){
  int index = x + (y * resolution.x);
   
  thrust::default_random_engine rng(hash(index*time));
  thrust::uniform_real_distribution<float> u01(0,1);

  return glm::vec3((float) u01(rng), (float) u01(rng), (float) u01(rng));
}

//TODO: IMPLEMENT THIS FUNCTION
//Function that does the initial raycast from the camera
__host__ __device__ ray raycastFromCameraKernel(glm::vec2 resolution, float time, int x, int y, glm::vec3 eye, glm::vec3 view, glm::vec3 up, glm::vec2 fov
#if DOF
											,float aperture, float focusPlane
#endif
	){
  ray r;

  // @DO: verify field of view!
  glm::vec3 axis_a = glm::cross(view, up);
  glm::vec3 axis_b = glm::cross(axis_a, view);
  glm::vec3 midPoint = eye + view;
  glm::vec3 viewPlaneX = axis_a * tan(PI_F * fov.x/180.0f) * glm::length(view)/glm::length(axis_a);
  glm::vec3 viewPlaneY = axis_b * tan(PI_F * fov.y/180.0f) * glm::length(view)/glm::length(axis_b);

#if JITTER
  glm::vec3 jitter = generateRandomNumberFromThread(resolution,time,x,y);
  glm::vec3 screenPoint = midPoint +
							(2.0f * ((jitter.x + 1.0f * x) / (resolution.x-1)) - 1.0f) * viewPlaneX + 
							(1.0f - 2.0f * ((jitter.y + 1.0f * y) / (resolution.y-1))) * viewPlaneY;
#else
  glm::vec3 screenPoint = midPoint +
							(2.0f * (1.0f * x / (resolution.x-1)) - 1.0f) * viewPlaneX + 
							(1.0f - 2.0f * (1.0f * y / (resolution.y-1))) * viewPlaneY;

#endif

#if DOF

  glm::vec3 focusPlaneIntersection;
  
  r.origin = eye;
  r.direction = glm::normalize(screenPoint - eye);

  glm::vec3 focusPlaneCenter = r.origin + r.direction * focusPlane;
  planeIntersectionTest(focusPlaneCenter,view,r,focusPlaneIntersection);

  glm::vec3 apertureJitter = aperture * (generateRandomNumberFromThread(resolution,time,x,y) - 0.5f);
  r.origin = r.origin + apertureJitter;
  r.direction = glm::normalize(focusPlaneIntersection - r.origin);

#else
  r.origin = screenPoint;
  r.direction = glm::normalize(screenPoint - eye);
#endif
  return r;
}

//Kernel that blacks out a given image buffer
__global__ void clearImage(glm::vec2 resolution, glm::vec3* image){
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int index = x + (y * resolution.x);
    if(x<=resolution.x && y<=resolution.y){
      image[index] = glm::vec3(0,0,0);
    }
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__ void sendImageToPBO(uchar4* PBOpos, glm::vec2 resolution, glm::vec3* image, float frames){
  
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);
  
  if(x<=resolution.x && y<=resolution.y){
      glm::vec3 color;
      color.x = image[index].x*255.0 / frames;
      color.y = image[index].y*255.0 / frames;
      color.z = image[index].z*255.0 / frames;

      if(color.x>255){
        color.x = 255;
      }

      if(color.y>255){
        color.y = 255;
      }

      if(color.z>255){
        color.z = 255;
      }
      
      // Each thread writes one pixel location in the texture (textel)
      PBOpos[index].w = 0;
      PBOpos[index].x = color.x;
      PBOpos[index].y = color.y;
      PBOpos[index].z = color.z;
  }
}

__device__ bool visibilityCheck(ray r, staticGeom* geoms, int numberOfGeoms, glm::vec3 pointToCheck, int geomShotFrom, int lightSourceIndex)
{
	bool visible = true;
	float distance = glm::length(r.origin - pointToCheck);

	// Check whether any object occludes point to check from ray's origin
	for(int iter=0; iter < numberOfGeoms; iter++)
	{
		// Avoid calculating self intersections
		if(iter==lightSourceIndex)
			continue;

		float depth=-1;
		glm::vec3 intersection;
		glm::vec3 normal;
		
		if(geoms[iter].type == CUBE)
		{
			depth = boxIntersectionTest(geoms[iter],r,intersection,normal);
		}
		
		
		else if(geoms[iter].type == SPHERE)
		{
			depth = sphereIntersectionTest(geoms[iter],r,intersection,normal);
		}
		
		if(depth > 0 && (depth + NUDGE) < distance)
		{
			//printf("Depth: %f\n", depth);
			visible = false;
			break;
		}
	}

	
	return visible;
}

//TODO: IMPLEMENT THIS FUNCTION
//Core raytracer kernel
__global__ void raytraceRay(glm::vec2 resolution, float time, cameraData cam, int rayDepth, glm::vec3* colors,
                            staticGeom* geoms, int numberOfGeoms, material* materials, int numberOfMaterials, ray* rayPool
#if COMPACTION
							, int numberOfRays
#endif
							){
	int index;
	int x,y;
#if COMPACTION
	index = blockIdx.x * blockDim.x + threadIdx.x;
	y = index / resolution.x;
	x = index - y * resolution.x;
	if (index < numberOfRays) {
#else
	x = (blockIdx.x * blockDim.x) + threadIdx.x;
	y = (blockIdx.y * blockDim.y) + threadIdx.y;
	index = x + (y * resolution.x);
	if((x<=resolution.x && y<=resolution.y) && glm::length(rayPool[index].transmission) > FLOAT_EPSILON){
#endif

	ray r = rayPool[index];	


	//Check all geometry for intersection
	int intersectedGeom = -1;
	int intersectedMaterial = -1;
	float minDepth = 1000000.0f;
	glm::vec3 minIntersectionPoint;
	glm::vec3 minNormal = glm::vec3(0.0f);
	for(int iter=0; iter < numberOfGeoms; iter++)
	{
		float depth=-1;
		glm::vec3 intersection;
		glm::vec3 normal;
		staticGeom currentGeometry = geoms[iter];
		if(currentGeometry.type == CUBE)
		{
			depth = boxIntersectionTest(currentGeometry,r,intersection,normal);
		}
		
		else if(geoms[iter].type == SPHERE)
		{
			depth = sphereIntersectionTest(currentGeometry,r,intersection,normal);
		}
		

		if(depth > 0 && depth < minDepth)
		{
			minDepth = depth;
			minIntersectionPoint = intersection;
			minNormal = normal;
			intersectedGeom = iter;
			intersectedMaterial = currentGeometry.materialid;
		}
	}

	// Depth render - test
	//float maxDepth = 15.0f;
	
	glm::vec3 diffuseLight = glm::vec3(0.0f);
	glm::vec3 phongLight = glm::vec3(0.0f);

	glm::vec3 diffuseColor;
	glm::vec3 specularColor;
	glm::vec3 emittance;

	//Calculate Lighting if any geometry is intersected
	if(intersectedGeom > -1)
	{
		//finalColor = materials[geoms[intersectedGeom].materialid].color;
		material m = materials[intersectedMaterial];
		diffuseColor = m.color;
		specularColor = m.specularColor;
		// Emmited color is the same as material color
		emittance = m.color * m.emittance;

#if PATHTRACER != 1
		// Stochastic Diffused Lighting with "area" lights
		for(int iter = 0; iter < numberOfGeoms; iter++)
		{
			material lightMaterial = materials[geoms[iter].materialid];
			// If this geometry is going to act like a light source
			if(lightMaterial.emittance > 0.0001f)
			{
				glm::vec3 lightSourceSample, normal;

				// Get a random point on the light source
				if(geoms[iter].type == SPHERE)
				{
					getRandomPointAndNormalOnSphere(geoms[iter],time*index, lightSourceSample, normal);
				}
				else if(geoms[iter].type == CUBE)
				{
					getRandomPointAndNormalOnCube(geoms[iter],time*index, lightSourceSample, normal);
				}

				// Diffuse Lighting Calculation
				glm::vec3 L = glm::normalize(lightSourceSample - minIntersectionPoint);
				
				//Shadow Ray check
				ray shadowRay;
				shadowRay.origin = minIntersectionPoint + NUDGE * L;
				shadowRay.direction = L;

				bool visible = visibilityCheck(shadowRay,geoms,numberOfGeoms,lightSourceSample, intersectedGeom, iter);

				if(visible)
				{
					diffuseLight += lightMaterial.color * lightMaterial.emittance * glm::max(glm::dot(L,minNormal),0.0f);

					
					// Calculate Phong Specular Part only if exponent is greater than 0
					if(m.specularExponent > FLOAT_EPSILON)
					{
						glm::vec3 reflectedLight = 2.0f * minNormal * glm::dot(minNormal, L) - L;
						phongLight += lightMaterial.color * lightMaterial.emittance * pow(glm::max(glm::dot(reflectedLight,minNormal),0.0f),m.specularExponent);
					}

				}
			}
		}
#endif;

		AbsorptionAndScatteringProperties absScatProps;
		glm::vec3 colorSend, unabsorbedColor;
		ray returnRay = r;
		int rayPropogation = calculateBSDF(returnRay,minIntersectionPoint,minNormal,diffuseColor*m.emittance,absScatProps,colorSend,unabsorbedColor,m);
		
		// Diffuse Reflection or light source
		if (rayPropogation == 0)
		{
#if PATHTRACER
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * emittance;
#else
			colors[index] += r.transmission * emittance;
#endif
#if PAINTERLY
			glm::vec3 randomVector = generateRandomNumberFromThread(resolution,time * (rayDepth+1),1,1);
#else
			glm::vec3 randomVector = generateRandomNumberFromThread(resolution,time * (rayDepth+1),x,y);
#endif
			r.direction = calculateRandomDirectionInHemisphere(minNormal, randomVector.x, randomVector.y);
			r.origin = minIntersectionPoint + 0.0005f * r.direction;
			r.transmission *= diffuseColor;
			
#else
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#else
			colors[index] += r.transmission * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#endif
			r.transmission = glm::vec3(0);
#endif
			rayPool[index] = r;

		}
		// Reflection; calculate transmission coeffiecient
		else if(rayPropogation == 1)
		{
#if PATHTRACER
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * emittance;
#else
			colors[index] += r.transmission * emittance;
#endif
#else
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * (1.0f - m.hasReflective) * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#else
			colors[index] += r.transmission * (1.0f - m.hasReflective) * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#endif
#endif
			returnRay.transmission = r.transmission * diffuseColor *  m.hasReflective;
			rayPool[index] = returnRay;
		}
		// Refraction; calculate transmission coeffiecient
		else if (rayPropogation == 2)
		{

#if PATHTRACER
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * emittance;
#else
			colors[index] += r.transmission * emittance;
#endif
#else
#if COMPACTION
			colors[r.pixelIndex] += r.transmission * (1.0f - m.hasRefractive) * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#else
			colors[index] += r.transmission * (1.0f - m.hasRefractive) * ( emittance + diffuseLight * diffuseColor +  phongLight * specularColor);
#endif
#endif
			returnRay.transmission = r.transmission * diffuseColor * m.hasRefractive;


#if FRESNEL
			// Fresnel Calculation

			// Fabs because the angle is always between 0 and 90, direction not-withstanding
			float nd = fabs(glm::dot(r.direction, minNormal));
			float nt = fabs(glm::dot(returnRay.direction, minNormal));
			float n_a = nd < 0 ? 1.0f : m.indexOfRefraction;
			float n_b = nd < 0 ? m.indexOfRefraction : 1.0f;
			float amountReflected;

#if SCHLICK
			// Schlick's Approximation

			float RO = (n_a - n_b) * (n_a - n_b) / ( (n_a + n_b) * (n_a + n_b));
			float c;
			if(n_a < n_b)
				c = 1 - nd;
			else
				c = 1 - nt;

			amountReflected = RO + (1-RO) * c * c * c * c * c;

#else
			// Fresnels equations
			float reflectedParallel = (n_b * nd - n_a * nt) * (n_b * nd - n_a * nt) / ((n_b * nd + n_a * nt) * (n_b * nd + n_a * nt));
			float reflectedPerpendicular = (n_a * nd - n_b * nt) * (n_a * nd - n_b * nt) / ((n_a * nd + n_b * nt) * (n_a * nd + n_b * nt));
			amountReflected = 0.5 * (reflectedParallel + reflectedPerpendicular);
#endif
			// Stochastically decide whether to reflect or refract
			glm::vec3 randVector = generateRandomNumberFromThread(resolution,time * (rayDepth+1),x,y);
			
			// If a uniform variable is less than the reflected amount, this ray shall be reflected
			if(randVector.y  < amountReflected)
			{
				returnRay.direction = r.direction - 2.0f * minNormal  * glm::dot(minNormal,r.direction);
				returnRay.origin = minIntersectionPoint + NUDGE * returnRay.direction;
			}
#endif
			rayPool[index] = returnRay;
		}
	}
	// No intersection, mark rays as dead
	// Ambeint term 
	else
	{
		glm::vec3 ambient = glm::vec3(0,0,0);
#if COMPACTION
		colors[r.pixelIndex] += ambient;
#else
		colors[index] += ambient; 
#endif
		r.transmission = glm::vec3(0.0f);
		rayPool[index] = r;
	}
	
	/*
		//Checking for correct ray direction
		colors[index].x = fabs(r.direction.x);
		colors[index].y = fabs(r.direction.y);
		colors[index].z = fabs(r.direction.z);
	
		//Check for correct material pickup
		colors[index] = color;

		//Checking for correct depth testing
		colors[index] = color * (maxDepth - minDepth)/maxDepth;

		//Checking for correct normals
		colors[index] = glm::vec3(minNormal);
		colors[index] = glm::vec3( fabs(minNormal.x), fabs(minNormal.y), fabs(minNormal.z));
	*/
	
   }
}

__global__ void fillRayPoolFromCamera(glm::vec2 resolution, float time, cameraData cam, ray* rayPool){

  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){

    
	ray r;
	r = raycastFromCameraKernel(resolution,time,x,y,cam.position,cam.view,cam.up,cam.fov
#if DOF
											,cam.aperture, cam.focusPlane
#endif
		);

#if COMPACTION
	r.pixelIndex = index;
#endif
	r.transmission = glm::vec3(1.0f);

	// Access global memory only once
	rayPool[index] = r;
   }
}

__global__ void combineIntoAccumulatorImage(glm::vec2 resolution, float frames, glm::vec3* inputColors, glm::vec3* displayColors)
{
  int x = (blockIdx.x * blockDim.x) + threadIdx.x;
  int y = (blockIdx.y * blockDim.y) + threadIdx.y;
  int index = x + (y * resolution.x);

  if((x<=resolution.x && y<=resolution.y)){
	  //displayColors[index] = (((frames-1) * displayColors[index]) + inputColors[index])/frames;

	  // Averaging done in SendToPBO
	  displayColors[index] += inputColors[index];
  }
}


#if COMPACTION

/***

	Compaction:
	- Mark whether ray should be compacted or not based on transmission
	- Inclusive Scan over predicate array (in place?) to generate output indices
	  Now all rays corresponding to an ouput index > 0 should go to index outputIndex - 1 in output array
	- Use a scatter operation to distribute ray data onto those locations in the output array.

***/

// NVidiaScan
__global__ void prescan(float *g_odata, float *g_idata, int n) 
{

	int bid = blockIdx.x;

	// must give final sum into auxilary array at bid.
	// maximum number of elements that can be prescanned?
	// number of threads per block * number of threads per block (each scan per block and then one block to scan all the in between things)
	// @TODO: how to scan more than the above described limit?

	// NVIDIA implenentation follows from: 
	// http://http.developer.nvidia.com/GPUGems3/gpugems3_ch39.html

	extern __shared__ float temp[];// allocated on invocation 
	int thid = threadIdx.x; 
	int offset = 1; 
	temp[2*thid] = g_idata[2*thid]; // load input into shared memory 
	temp[2*thid+1] = g_idata[2*thid+1]; 
	for (int d = n>>1; d > 0; d >>= 1) // build sum in place up the tree 
	{ 
		__syncthreads(); 
		if (thid < d) 
		{ 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			temp[bi] += temp[ai]; 
		} 
		offset *= 2; 
	} 
	if (thid == 0) { temp[n - 1] = 0; } // clear the last element 
	for (int d = 1; d < n; d *= 2) // traverse down tree & build scan 
	{ 
		offset >>= 1; 
		__syncthreads(); 
		if (thid < d) 
		{ 
			int ai = offset*(2*thid+1)-1; 
			int bi = offset*(2*thid+2)-1; 
			float t = temp[ai]; 
			temp[ai] = temp[bi]; 
			temp[bi] += t; 
		} 
	} 
	__syncthreads(); 
	g_odata[2*thid] = temp[2*thid]; // write results to device memory 
	g_odata[2*thid+1] = temp[2*thid+1]; 
}

// Mark with predicate whether active or inactive
__global__ void predicateMark(ray* inputRays, int* outputPredicate, int size)
{
	// Using 1D kernel for compaction
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;

	if(index < size)
	{
		if(glm::length(inputRays[index].transmission) > FLOAT_EPSILON)
		{
			outputPredicate[index] = 1;
		}
		else
		{
			outputPredicate[index] = 0;
		}
	}
}

// Scan Per Block
__global__ void naiveScanPerBlock(int *inData, int* outData, int *blockSum, int size)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	int threadId = threadIdx.x;

	// third arg of kernel launch = numFloats*sizeofFloat
	extern __shared__ int sharedElements[];

	// Fill elements into shared Memory
	if(index < size)
		sharedElements[threadId] = inData[index];
	// If last block, fill in zeros for padding
	else
		sharedElements[threadId] = 0;

	__syncthreads();

	int numSteps = ceil(log(1.0f*blockDim.x)/log(2.0f));
	int offset = 1;
	for(int d = 0; d < numSteps; d++, offset *= 2)
	{
		int t = sharedElements[threadId];
		if(threadId >= offset)
		{
			t = t + sharedElements[threadId - offset];
		}
		__syncthreads();
		
		sharedElements[threadId] = t;
		__syncthreads();
	}
	
	outData[index] = sharedElements[threadId];
	if(threadIdx.x == blockDim.x-1)
	{
		blockSum[blockIdx.x] = sharedElements[threadId];
	}
}

// Add back Scanned blocksums
__global__ void addBackBlockSums(int *outData, int *blockSum, int size, int *returnedSum)
{
	int index = blockIdx.x * blockDim.x  +threadIdx.x;
	if(blockIdx.x > 0)
		outData[index] = outData[index] + blockSum[blockIdx.x-1];
	if(index == size - 1)
		returnedSum[0] = outData[index];
}

// Wrap around total scanned array
int parallelScan(int *inData, int *outData, int size, int d=0)
{
	if(size==0)
		return 0;

	int threads = 1024;
	int blocks = ceil(size*1.0f/threads);
	dim3 blocksPerGrid(blocks,1,1);
	dim3 threadsPerBlock(threads,1,1);

	//std::cout<<"ParallelScan in "<<blocks<<" blocks of "<<threads<<" threads\n";

	int *cudaBlockSum;
	hipMalloc((void**)&cudaBlockSum, blocks*sizeof(int));

	int *cudaBlockSumScan;
	hipMalloc((void**)&cudaBlockSumScan, blocks*sizeof(int));

	int *sum;
	sum = (int*)malloc(sizeof(int));
	sum[0] = 0;
	int *cudaSum;
	hipMalloc((void**)&cudaSum,sizeof(int));

	naiveScanPerBlock<<<blocksPerGrid, threadsPerBlock, threads*sizeof(int)>>>(inData, outData,cudaBlockSum,size);
	checkCUDAError("Naive Scan Failed!");	

	// If the number of blocks is 1
	//if()
	//	only do naive scan per block for one block and copy into cudaBlockSumScan
	//	naiveScanPerBlock<<<1,threadsPerBlock, threads*sizeof(float)>>>(....)
	//else			recursive
	//	parallelScan(cudaBlockSum,cudaBlockSumScan,blocks);

	// Base Case of recursion
	if (blocks==1)
	{
		hipMemcpy(cudaSum,cudaBlockSum,sizeof(int),hipMemcpyDeviceToDevice);
		//return 0;
	}
	else
	{
		parallelScan(cudaBlockSum,cudaBlockSumScan,blocks,d+1);
		addBackBlockSums<<<blocksPerGrid, threadsPerBlock>>>(outData, cudaBlockSumScan, size, cudaSum);
		checkCUDAError("Add Back Blocks Failed!");	
	}
	

	if(d==0)
	{
		hipMemcpy(sum,cudaSum,sizeof(int),hipMemcpyDeviceToHost);
	}

	hipFree(cudaSum);
	hipFree(cudaBlockSumScan);
	hipFree(cudaBlockSum);
	return (*sum);

}

// Scatter Rays into the appropriate locations in the output array
__global__ void scatter(ray* inputRays, ray* outputRays, int* predicate, int* scatterIndices, int size)
{
	// Using 1D kernel for compaction
	int index = (blockIdx.x * blockDim.x) + threadIdx.x;
	
	if(index < size)
	{
		int scatterIndex = scatterIndices[index];
		// Data with scatter index < 0 is data not to be used
		if(predicate[index] > 0)
		{
			outputRays[scatterIndex - 1] = inputRays[index];
		}
	}
}

// Stream Compaction
int streamCompactRayPool(ray* inputRays, ray* outputRays, int size)
{

	/*
	// Testing recursive scan primitive

	int inputData[] = {1,1,1,1,1,1,1,1};//{1,1,2,3,4,5,6,7};
	int dataSize = 8;

	int *cudaInputData;
	hipMalloc((void**)&cudaInputData,dataSize*sizeof(int));
	hipMemcpy(cudaInputData,inputData,dataSize*sizeof(int),hipMemcpyHostToDevice);

	int *cudaOutputData;
	hipMalloc((void**)&cudaOutputData,dataSize*sizeof(int));

	int val = parallelScan(cudaInputData,cudaOutputData,dataSize);


	std::cout<<"Number of elements: "<<val<<std::endl;

	int *outputData;
	outputData = (int*)malloc(dataSize*sizeof(int));
	hipMemcpy(outputData,cudaOutputData,dataSize*sizeof(int),hipMemcpyDeviceToHost);

	for(int i=0; i<dataSize; i++)
		std::cout<<outputData[i]<<", ";

	std::cout<<std::endl;

	//free(inputData);
	free(outputData);
	hipFree(cudaInputData);
	hipFree(cudaOutputData);

	getchar();

	*/

	int* predicateArray;
	hipMalloc((void**)&predicateArray,size*sizeof(int));

	int* scatterLocations;
	hipMalloc((void**)&scatterLocations,size*sizeof(int));


	int numThreads = 512;
	int numBlocks = ceil(size*1.0f/numThreads);

	// Mark Predicates
	predicateMark<<<dim3(numBlocks,1,1),dim3(numThreads,1,1)>>>(inputRays, predicateArray, size);
	checkCUDAError("Predicate Mark Failed!");	

	// Scan Predicate to get location and also total number of final rays
	int compactedSize = parallelScan(predicateArray,scatterLocations,size);

	// Scatter rays to new locations in output array
	scatter<<<dim3(numBlocks,1,1),dim3(numThreads,1,1)>>>(inputRays,outputRays,predicateArray,scatterLocations,size);
	checkCUDAError("Scatter Failed!");	

	hipFree(scatterLocations);
	hipFree(predicateArray);
	
	return compactedSize;
}


#endif



#if PHOTONMAP

// Create a helper function to call these functions

//function for emitting photons from a sphere light
__global__ void emitPhotons(photon* photonPool, int numPhotons, int numBounces, staticGeom* geoms, int* lights, int numberOfLights,
														float* cudaAccumLightProbabilities, material* materials, float time)
{
	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if(index < numPhotons)
	{
		photon p;
		
		// Do a random-check to choose a certain light: take into consideration area of lights
		// Do a better random generation
		glm::vec3 randoms = generateRandomNumberFromThread(glm::vec2(800,800),time,index,numberOfLights);
		
		// Pick light based on cudaAccumLightProbabilities
		int lightIndex;
		bool largerThanPrev = true;			//whether randoms.x is larger than the previous probability in cudaAccumLightProbabilities
		for (int i=0; i<numberOfLights; ++i) {
			if (randoms.x <= cudaAccumLightProbabilities[i]) {
				if (largerThanPrev) {
					lightIndex = i;
					break;
				}
				else {
					largerThanPrev = true;
				}
			}
		}

		staticGeom lightChosen = geoms[lights[lightIndex]];			//get the light using the index from the lights array
		
		// for now only supports sphere and cube lights
		glm::vec3 position, normal;
		if(lightChosen.type == SPHERE)
		{
			getRandomPointAndNormalOnSphere(lightChosen,index, position, normal);
		}
		else if (lightChosen.type == CUBE)
		{
			getRandomPointAndNormalOnCube(lightChosen,index, position, normal);
		}
		p.position = position;

		// Shooting direction is normal at the point or random direction?
		// I think the lecture said choose random direction.
		p.dout = calculateRandomDirectionInHemisphere(normal,randoms.y,randoms.z);
		p.din = glm::vec3(0.0f);
		
		
		// Set color of photon
		material lightMaterial = materials[lightChosen.materialid];
		p.color = lightMaterial.emittance * lightMaterial.color;

		// Set whether photon has been stored/absorbed (dead)
		p.stored = false;
		p.geomid = lights[lightIndex];
		p.bounces = 0;		//increment the number of bounces by 1

		photonPool[index] = p;

		//set the rest of the photons in the array to not stored
		for (int i = 1; i < numBounces; ++i) {
			photon placeHolder;
			placeHolder.color = glm::vec3(0.0f);
			placeHolder.stored = false;
			placeHolder.bounces = -1;
			photonPool[numPhotons * i + index] = placeHolder;
		}
	}

}

__global__ void displayPhotons(photon* photonPool, int numPhotons, int numBounces, glm::vec2 resolution, cameraData cam, glm::vec3* colors, cudaMat4 viewProjectionViewport, float flux)
{

	int index = blockIdx.x * blockDim.x + threadIdx.x;
	
	if(index < numPhotons)
	{
		
		for (int i = 0; i < numBounces; ++i) {
		
			photon p = photonPool[numPhotons * i + index];

			//only display color if photon is not dead at that position
			if (p.stored) {
				glm::vec3 photonToEye = cam.position - p.position;
				
				// Do this assuming view, projection and viewport matrices are provided
				//glm::vec3 screenPosition = multiplyMV(viewProjectionViewport,glm::vec4(p.position,1.0f));
				
				glm::vec4 screenPosition = multiplyMV_4(viewProjectionViewport, glm::vec4(p.position, 1.0f));

				// Shift to viewport matrix
				//transform to clip
				screenPosition.x /= screenPosition.w;
				screenPosition.y /= screenPosition.w;
				screenPosition.z /= screenPosition.w;

				//transform to screen coord
				screenPosition.x = (screenPosition.x+1) * resolution.x/2.0f;
				screenPosition.y = (-screenPosition.y+1) * resolution.y/2.0f;

				if(screenPosition.x >=0 && screenPosition.x < resolution.x && screenPosition.y >=0 && screenPosition.y < resolution.y)
				{
					// write to the color buffer!
					// race conditions?
					int x = screenPosition.x;
					int y = screenPosition.y;
					int pixelIndex = x + (y * resolution.x);
					//colors[pixelIndex] = glm::abs(p.dout);		//glm::abs causes a kernel failure on my computer...
					colors[pixelIndex] = p.color;
				}
			}

		}
	}
}

__global__ void testImage(glm::vec3* colors, glm::vec2 resolution) {
	
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);

	if (x <= resolution.x && y <= resolution.y) { 
		colors[index] = glm::vec3(1.0);
	}

}

__global__ void bouncePhotons(photon* photonPool, int numPhotons, int currentBounces, staticGeom* geoms, int numberOfGeoms, material* materials, float time)
{
	//bounce photons around
	int index = blockIdx.x * blockDim.x + threadIdx.x; 

	if (index < numPhotons){

		int prevIndex = index;
		if (currentBounces!=0)
			prevIndex = index + (currentBounces-1) * numPhotons;

		int nextIndex = index + currentBounces * numPhotons;

		//load a photon from memory
		photon p = photonPool[prevIndex];

		//create ray using photon
			ray r;
			r.origin = p.position + 0.01f*p.dout;		//offset point a little to avoid self intersection
			r.direction = p.dout;

			//intersection testing
			int intersectedGeom = -1;
			int intersectedMaterial = -1;
			float minDepth = 1000000.0f;
			glm::vec3 minIntersectionPoint;
			glm::vec3 minNormal = glm::vec3(0.0f);
			
			for (int iter=0; iter < numberOfGeoms; iter++)
			{
					float depth=-1;
					glm::vec3 intersection;
					glm::vec3 normal;
					staticGeom currentGeometry = geoms[iter];
					if (currentGeometry.type == CUBE)
					{
							depth = boxIntersectionTest(currentGeometry,r,intersection,normal);
					}
					
					else if (geoms[iter].type == SPHERE)
					{
							depth = sphereIntersectionTest(currentGeometry,r,intersection,normal);
					}
					

					if (depth > 0 && depth < minDepth)
					{
							minDepth = depth;
							minIntersectionPoint = intersection;
							minNormal = normal;
							intersectedGeom = iter;
							intersectedMaterial = currentGeometry.materialid;
					}
			}

			//if intersection occurs, accumulate color and keep bouncing
			if (intersectedGeom > -1) {

				material m = materials[intersectedMaterial];

				//assume diffuse surfaces only for now, so bounce in random direction
				p.color *= m.color;

				glm::vec3 randoms = generateRandomNumberFromThread(glm::vec2(800,800),time,index,currentBounces+1);
				p.din = p.dout;
				//p.stored = true;
				p.dout = calculateRandomDirectionInHemisphere(minNormal,randoms.y,randoms.z);
				p.position = minIntersectionPoint;

				AbsorptionAndScatteringProperties absScatProps;
				glm::vec3 colorSend, unabsorbedColor;
				ray returnRay = r;
				
				int rayPropogation = calculateBSDF(returnRay,minIntersectionPoint,minNormal,p.color,absScatProps,colorSend,unabsorbedColor,m);

				// Reflection; calculate transmission coeffiecient
				if(rayPropogation == 1)
				{
					p.dout = returnRay.direction;
					p.color = p.color * m.hasReflective;
					p.stored = true;
				}
				// Refraction; calculate transmission coeffiecient
				else if (rayPropogation == 2)
				{
					p.color = p.color * m.hasRefractive;
					p.stored = true;

#if FRESNEL
					// Fresnel Calculation

					// Fabs because the angle is always between 0 and 90, direction not-withstanding
					float nd = fabs(glm::dot(r.direction, minNormal));
					float nt = fabs(glm::dot(returnRay.direction, minNormal));
					float n_a = nd < 0 ? 1.0f : m.indexOfRefraction;
					float n_b = nd < 0 ? m.indexOfRefraction : 1.0f;
					float amountReflected;

#if SCHLICK
					// Schlick's Approximation

					float RO = (n_a - n_b) * (n_a - n_b) / ( (n_a + n_b) * (n_a + n_b));
					float c;
					if(n_a < n_b)
						c = 1 - nd;
					else
						c = 1 - nt;

					amountReflected = RO + (1-RO) * c * c * c * c * c;

#else
					// Fresnels equations
					float reflectedParallel = (n_b * nd - n_a * nt) * (n_b * nd - n_a * nt) / ((n_b * nd + n_a * nt) * (n_b * nd + n_a * nt));
					float reflectedPerpendicular = (n_a * nd - n_b * nt) * (n_a * nd - n_b * nt) / ((n_a * nd + n_b * nt) * (n_a * nd + n_b * nt));
					amountReflected = 0.5 * (reflectedParallel + reflectedPerpendicular);
#endif
					// Stochastically decide whether to reflect or refract
					glm::vec3 randVector = generateRandomNumberFromThread(glm::vec2(637,791),time,index,currentBounces+1);

					// If a uniform variable is less than the reflected amount, this ray shall be reflected
					if(randVector.y  < amountReflected)
					{
						p.dout = r.direction - 2.0f * minNormal  * glm::dot(minNormal,r.direction);
					}
					else
					{
						p.dout = returnRay.direction;
					}
#endif

				}
				// Default to diffuse
				else
				{
					p.stored = true;
				}
				
				p.geomid = intersectedGeom;
			}
			else {
				//kill the photon if it doesn't intersect with anything
				//When using stream compaction, need to figure if photon is stored or dead or (alive and kicking)
				p.stored = false;
				p.geomid = -1;
			}

			p.bounces ++;
			//write new bounced photon into memory
			
			photonPool[nextIndex] = p;
		}

}

#define oneOverSqrtTwoPi 0.3989422804f
__device__ float gaussianWeight( float dx, float radius)
{
	float sigma = radius/3.0;
	return (oneOverSqrtTwoPi / sigma) * exp( - (dx*dx) / (2.0 * sigma * sigma) );
}

// Caculate radiances from photons
__global__ void gatherPhotons(glm::vec2 resolution, float time, cameraData cam, glm::vec3* colors, staticGeom* geoms,
															int numberOfGeoms, ray* rayPool, photon* photons, int numPhotons, int numBounces, float flux) {

	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;
	int index = x + (y * resolution.x);
	if((x<=resolution.x && y<=resolution.y) && glm::length(rayPool[index].transmission) > FLOAT_EPSILON){
		ray r = rayPool[index];	

		//Check all geometry for intersection
		int intersectedGeom = -1;
		int intersectedMaterial = -1;
		float minDepth = 1000000.0f;
		glm::vec3 minIntersectionPoint;
		glm::vec3 minNormal = glm::vec3(0.0f);
		for(int iter=0; iter < numberOfGeoms; iter++)
		{
				float depth=-1;
				glm::vec3 intersection;
				glm::vec3 normal;
				staticGeom currentGeometry = geoms[iter];
				if(currentGeometry.type == CUBE)
				{
					depth = boxIntersectionTest(currentGeometry,r,intersection,normal);
				}
		
				else if(geoms[iter].type == SPHERE)
				{
					depth = sphereIntersectionTest(currentGeometry,r,intersection,normal);
				}
		

				if(depth > 0 && depth < minDepth)
				{
					minDepth = depth;
					minIntersectionPoint = intersection;
					minNormal = normal;
					intersectedGeom = iter;
					intersectedMaterial = currentGeometry.materialid;
				}

		}

		//Calculate radiance if any geometry is intersected
		if(intersectedGeom > -1)
		{
			glm::vec3 accumColor(0);
			//Use brute force search to find the photons that are within a certain radius
			for (int i=0; i<numPhotons * numBounces; ++i) {
				photon p = photons[i];
				float photonDistance  = glm::distance(minIntersectionPoint, p.position);
				// Indirect Illumination only?
				if ( photonDistance <= RADIUS && p.geomid == intersectedGeom && p.bounces > 0) {
					//Is lambert brdf cos(theta_i)?
					accumColor += gaussianWeight(photonDistance, RADIUS) *  max(0.0f, glm::dot(minNormal, -p.din)) * p.color;
				}
			}
			colors[index] += accumColor * flux;
		}
	}
}


void tracePhotons(int photonThreadsPerBlock, int photonBlocksPerGrid, photon* cudaPhotonPool,
									int numPhotons, staticGeom* cudaGeoms, int numberOfGeoms, material* cudaMaterials,
									float time)
{
	for(int i=0; i < numBounces; i++)
	{
		// Bounce Photons Around
		bouncePhotons<<<dim3(photonBlocksPerGrid),dim3(photonThreadsPerBlock)>>>(cudaPhotonPool, numPhotons, i,	cudageoms, numGeoms, cudamaterials, time);

#if COMPACTION
		// Do some compaction

#endif
	}
}


void initPhotonMap()
{
	//Create Memory for RayPool
	cudaPhotonPool = NULL;
	hipMalloc((void**)&cudaPhotonPool, numBounces * numPhotons * sizeof(photon));
	
}

void cleanPhotonMap()
{
	hipFree(cudaPhotonPool);
}

//allocate memory for geometry data
void cudaAllocateMemory(camera* renderCam, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms) {

	int size = (int)renderCam->resolution.x*(int)renderCam->resolution.y;
	
	//send image to GPU
	cudaimage = NULL;
	hipMalloc((void**)&cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	hipMemcpy( cudaimage, renderCam->image, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyHostToDevice);
	
	//package geometry and materials and sent to GPU
	staticGeom* geomList = new staticGeom[numberOfGeoms];
	std::vector<int> lightVec;
	numGeoms = numberOfGeoms;

	//get geom from frame 0
	for(int i=0; i<numberOfGeoms; i++){
			staticGeom newStaticGeom;
			newStaticGeom.type = geoms[i].type;
			newStaticGeom.materialid = geoms[i].materialid;
			newStaticGeom.translation = geoms[i].translations[0];
			newStaticGeom.rotation = geoms[i].rotations[0];
			newStaticGeom.scale = geoms[i].scales[0];
			newStaticGeom.transform = geoms[i].transforms[0];
			newStaticGeom.inverseTransform = geoms[i].inverseTransforms[0];
			geomList[i] = newStaticGeom;

			//store which objects are lights
			if(materials[geoms[i].materialid].emittance > 0)
					lightVec.push_back(i);
	}

	cudageoms = NULL;
	hipMalloc((void**)&cudageoms, numberOfGeoms*sizeof(staticGeom));
	hipMemcpy( cudageoms, geomList, numberOfGeoms*sizeof(staticGeom), hipMemcpyHostToDevice);

	//copy materials to memory
	cudamaterials = NULL;
	hipMalloc((void**)&cudamaterials, numberOfMaterials*sizeof(material));
	hipMemcpy( cudamaterials, materials, numberOfMaterials*sizeof(material), hipMemcpyHostToDevice);

	//copy light ID to memeory
	numLights = lightVec.size();

	int* lightID = new int[numLights];
	for(int i = 0; i <numLights; ++i)
			lightID[i] = lightVec[i];

	cudaLights = NULL;
	hipMalloc((void**)&cudaLights, numLights*sizeof(int));
	hipMemcpy( cudaLights, lightID, numLights*sizeof(int), hipMemcpyHostToDevice);

#if PHOTONMAP
	//std::cout<<"allocating mem for photon pool"<<std::endl;
	cudaPhotonPool = NULL;
	hipMalloc((void**)&cudaPhotonPool, numBounces * numPhotons * sizeof(photon));

	cudaPhotonMapImage = NULL;
	hipMalloc((void**)&cudaPhotonMapImage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));

	// compute the accumulated probablity of photons being emitted from each light
	float totalEmittanceTimesArea = 0;
	float* accumAccumLightProbabilities = new float[numLights];
	for (int i=0; i<numLights; ++i) {
		staticGeom light = geomList[lightID[i]];
		material lightmtl = materials[light.materialid];

		float area;
		if (light.type == SPHERE) {
			// compute the surface area of an ellipsoid
			float a = 0.5 * light.scale.x;
			float b = 0.5 * light.scale.y;
			float c = 0.5 * light.scale.z;
			float p = 1.6075;
			float ap = pow(a, p);
			float bp = pow(b, p);
			float cp = pow(c, p);
			area = 4 * PI_F * pow((ap*bp + ap*cp + bp*cp)/3, 1/p); //the approximate formula for surface area of ellipsoids
		}
		else if (light.type == CUBE) {
			// compute the surface area of a box
			float sx = light.scale.x;
			float sy = light.scale.y;
			float sz = light.scale.z;
			area = 2 * sx * sy + 2 * sx * sz + 2 * sy * sz;
		}
		totalEmittanceTimesArea += area * lightmtl.emittance;
		accumAccumLightProbabilities[i] = totalEmittanceTimesArea;
	}

	// divide accumulated "emittance * area" of each light by totalEmittanceTimesArea to get the probablity of being emitted from each light
	for (int i=0; i<numLights; ++i) {
		accumAccumLightProbabilities[i] /= totalEmittanceTimesArea;
	}

	cudaAccumLightProbabilities = NULL;
	hipMalloc((void**)&cudaAccumLightProbabilities, numLights*sizeof(float));
	hipMemcpy( cudaAccumLightProbabilities, accumAccumLightProbabilities, numLights*sizeof(float), hipMemcpyHostToDevice);
#endif

	cudaAllocateAccumulatorImage(renderCam);

	delete[] geomList;
	delete[] lightID;
}

//free up memory
void cudaFreeMemory() {

	hipFree( cudaimage);
	hipFree( cudageoms );
	hipFree( cudamaterials);
	hipFree( cudaLights);

#if PHOTONMAP
	hipFree(cudaPhotonPool);
	hipFree(cudaPhotonMapImage);
#endif

	cudaFreeAccumulatorImage();

}


void cudaPhotonMapCore(camera* renderCam, int frame, int iterations, uchar4* PBOpos, cameraData liveCamera)
{

	// Set up crucial magic
	glm::vec2 resolution = renderCam->resolution;
	int tileSize = 8;
	dim3 pixelThreadsPerBlock(tileSize, tileSize);
	dim3 pixelBlocksPerGrid((int)ceil(float(resolution.x)/float(tileSize)), (int)ceil(float(resolution.y)/float(tileSize)));

	//package camera data
	cameraData cam;
	cam.resolution = renderCam->resolution;
	cam.position = renderCam->positions[frame];
	cam.view = renderCam->views[frame];
	cam.up = renderCam->ups[frame];
	cam.fov = renderCam->fov;
	cam.focusPlane = renderCam->focusPlanes[frame];
	cam.aperture = renderCam->apertures[frame];

	//user interaction
	cam.position +=  (liveCamera.position);
	cam.view = glm::normalize(cam.view + liveCamera.view);
	cam.aperture += liveCamera.aperture;
	cam.focusPlane += liveCamera.focusPlane;

	// Clear photon image buffer
	clearImage<<<pixelBlocksPerGrid,pixelThreadsPerBlock>>>(resolution, cudaPhotonMapImage);
	hipDeviceSynchronize();
	checkCUDAError("clearImage kernel failed!");

	// Generate Photon List
	int photonThreadsPerBlock = 512;
	int photonBlocksPerGrid = ceil(numPhotons * 1.0f/photonThreadsPerBlock);

	emitPhotons<<<dim3(photonBlocksPerGrid),dim3(photonThreadsPerBlock)>>>(cudaPhotonPool, numPhotons, numBounces, cudageoms, 
		cudaLights, numLights, cudaAccumLightProbabilities, cudamaterials, iterations);
	hipDeviceSynchronize();
	checkCUDAError("emit photons kernel failed!");

	// Trace all photons with all bounces
	tracePhotons(photonThreadsPerBlock, photonBlocksPerGrid, cudaPhotonPool, numPhotons, cudageoms, numGeoms, cudamaterials, iterations);
	hipDeviceSynchronize();
	checkCUDAError("tracePhotons kernel failed!");

	// Assume each light emits the same number of photons, calculate the flux per photon
	float flux = totalEnergy/((float)numPhotons/(float)numLights);


if (mode == DISP_GATHER)
{
	// Compute radiance from photons
	// Generate rays first
	ray* cudarays = NULL;
	hipMalloc((void**)&cudarays, (renderCam->resolution.x * renderCam->resolution.y) * sizeof(ray));
	fillRayPoolFromCamera<<<pixelBlocksPerGrid, pixelThreadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, cudarays);
	hipDeviceSynchronize();
	checkCUDAError("fill ray pool kernel failed!");

	// Assume each light emits the same number of photons, calculate the flux per photon
	float flux = totalEnergy/(float)numPhotons;

	gatherPhotons<<<pixelBlocksPerGrid, pixelThreadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, cudaPhotonMapImage, cudageoms, numGeoms,
		cudarays, cudaPhotonPool, numPhotons, numBounces, flux);
	hipDeviceSynchronize();
	checkCUDAError("gather photonskernel failed!");

	hipFree(cudarays);
	hipDeviceSynchronize();
	checkCUDAError("free ray pool failed!");
}
else if (mode == DISP_PHOTONS)
{
	// Calculate Viewport * Projection * View matrix from camera info
	glm::vec3 center = cam.position + cam.view;

	glm::mat4 viewMat = glm::lookAt(cam.position, center, cam.up);
	glm::mat4 projectionMat = glm::perspective(cam.fov.y*2, cam.resolution.x/cam.resolution.y, 0.1f, 1000.0f);

	cudaMat4 viewProjectionViewPort = utilityCore::glmMat4ToCudaMat4(projectionMat*viewMat);
	
	//utilityCore::printCudaMat4(viewProjectionViewPort);

	// Display all photons in the photonImage buffer
	displayPhotons<<<dim3(photonBlocksPerGrid),dim3(photonThreadsPerBlock)>>>(cudaPhotonPool, numPhotons, numBounces, resolution, 
		cam, cudaPhotonMapImage, viewProjectionViewPort, flux);
	hipDeviceSynchronize();
	checkCUDAError("display photons kernel failed!");
}


#if ACCUMULATION
	combineIntoAccumulatorImage<<<pixelBlocksPerGrid,pixelThreadsPerBlock>>>(renderCam->resolution, (float)iterations, cudaPhotonMapImage, accumulatorImage);
	sendImageToPBO<<<pixelBlocksPerGrid,pixelThreadsPerBlock>>>(PBOpos, renderCam->resolution, accumulatorImage, (float)iterations);
#else
	sendImageToPBO<<<pixelBlocksPerGrid,pixelThreadsPerBlock>>>(PBOpos, renderCam->resolution, cudaPhotonMapImage, 1.0f);
#endif
	hipDeviceSynchronize();
	checkCUDAError("Send to PBO kernel failed!");

	//retrive image from GPU
	int imageSize = (int)resolution.x * (int) resolution.y;
#if ACCUMULATION
	hipMemcpy(renderCam->image, accumulatorImage, imageSize*sizeof(glm::vec3), hipMemcpyDeviceToHost);
#else
	hipMemcpy(renderCam->image, cudaPhotonMapImage, imageSize*sizeof(glm::vec3), hipMemcpyDeviceToHost);
#endif

	hipDeviceSynchronize();
	checkCUDAError("Photon mapping kernel failed!");

}
#endif	

void cudaRaytraceCore(uchar4* PBOpos, camera* renderCam, int frame, int iterations, material* materials, int numberOfMaterials, geom* geoms, int numberOfGeoms, cameraData liveCamera){

  // testing 
  //streamCompact();

  //if(iterations == 0)
  //{
  //  // Allocate Accumulator Image
  //  cudaAllocateAccumulatorImage(renderCam);
  //}

  int traceDepth = 1; //determines how many bounces the raytracer traces

  // set up crucial magic
  int tileSize = 8;
  dim3 threadsPerBlock(tileSize, tileSize);
  dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

  //package camera
  cameraData cam;
  cam.resolution = renderCam->resolution;
  cam.position = renderCam->positions[frame];
  cam.view = renderCam->views[frame];
  cam.up = renderCam->ups[frame];
  cam.aperture = renderCam->apertures[frame];
  cam.focusPlane = renderCam->focusPlanes[frame];
  cam.fov = renderCam->fov;

  //user interaction
  cam.position +=  (liveCamera.position);
  cam.view = glm::normalize(cam.view + liveCamera.view);
  cam.aperture += liveCamera.aperture;
  cam.focusPlane += liveCamera.focusPlane;

  //Create Memory for RayPool
  ray* cudarays = NULL;
  hipMalloc((void**)&cudarays, (renderCam->resolution.x * renderCam->resolution.y) * sizeof(ray));

#if COMPACTION
  ray* cudarays2 = NULL;
  hipMalloc((void**)&cudarays2, (renderCam->resolution.x * renderCam->resolution.y) * sizeof(ray));
#endif

  //clear On screen buffer
  clearImage<<<fullBlocksPerGrid,threadsPerBlock>>>(renderCam->resolution, cudaimage);

  //Fill ray pool with rays from camera for first iteration
  fillRayPoolFromCamera<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, cudarays);
  int numberOfRays = (int)renderCam->resolution.x * (int)renderCam->resolution.y;

  //std::cout<<"StreamCompaction: ";

  int linearTileSize = tileSize*tileSize;
  for(int i=0; i < MAX_RECURSION_DEPTH && numberOfRays > 0; i++)
  {
#if COMPACTION
	    dim3 linearGridSize((int)ceil(numberOfRays*1.0f/linearTileSize),1,1);
		raytraceRay<<<linearGridSize, dim3(linearTileSize,1,1)>>>(renderCam->resolution, (float)iterations, cam, traceDepth+i,
															cudaimage, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials,
															i%2==0?cudarays : cudarays2,
															numberOfRays);
		checkCUDAError("Ray Trace Failed!");	 

 		numberOfRays = streamCompactRayPool( i%2==0? cudarays : cudarays2,
			  						         i%2==0? cudarays2 : cudarays,
										     numberOfRays);
#else
	//kernel launches
	raytraceRay<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cam, traceDepth+i, cudaimage, cudageoms, numberOfGeoms, cudamaterials, numberOfMaterials, cudarays);
#endif
  }

  //getchar();

#if ACCUMULATION
  combineIntoAccumulatorImage<<<fullBlocksPerGrid, threadsPerBlock>>>(renderCam->resolution, (float)iterations, cudaimage, accumulatorImage);
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, accumulatorImage, (float)iterations);
#else
  sendImageToPBO<<<fullBlocksPerGrid, threadsPerBlock>>>(PBOpos, renderCam->resolution, cudaimage, 1.0f);
#endif

  //retrieve image from GPU for sending to bmp file
  if(singleFrameMode)
#if ACCUMULATION
	hipMemcpy( renderCam->image, accumulatorImage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
#else
	hipMemcpy( renderCam->image, cudaimage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3), hipMemcpyDeviceToHost);
#endif

  //free up stuff, or else we'll leak memory like a madman
  hipFree( cudarays );
#if COMPACTION
  hipFree( cudarays2);
#endif
  // make certain the kernel has completed
  hipDeviceSynchronize();

  checkCUDAError("Kernel failed!");
}


//Clear AccumulatorImage. For an interactive application, this needs to be called everytime the camera moves or the scene changes
void cudaClearAccumulatorImage(camera *renderCam)
{
	// set up crucial magic
  	int tileSize = 8;
	dim3 threadsPerBlock(tileSize, tileSize);
    dim3 fullBlocksPerGrid((int)ceil(float(renderCam->resolution.x)/float(tileSize)), (int)ceil(float(renderCam->resolution.y)/float(tileSize)));

	clearImage<<<fullBlocksPerGrid,threadsPerBlock>>>(renderCam->resolution, accumulatorImage);
}

//Allocate Memory For AccumulatorImage
void cudaAllocateAccumulatorImage(camera *renderCam)
{
	hipMalloc((void**)&accumulatorImage, (int)renderCam->resolution.x*(int)renderCam->resolution.y*sizeof(glm::vec3));
	cudaClearAccumulatorImage(renderCam);
}

//Free memory of the accumulator image
void cudaFreeAccumulatorImage()
{
	hipFree(accumulatorImage);
}